
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

__global__ void monteCarloEuler(int *count, int N, unsigned long long seed) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    int local_k = 0;

    // Divida o total de iterações entre todas as threads
    for (int i = tid; i < N; i += total_threads) {
        double x = hiprand_uniform_double(&state);
        double y = hiprand_uniform_double(&state);

        // Verificando se está dentro do círculo
        if (x * x + y * y <= 1.0) {
            local_k++;
        }
    }

    // Soma local_k em um único valor usando atomicAdd
    atomicAdd(count, local_k);
}

int main() {
    // Configurando a precisão do resultado
    std::cout << std::setprecision(15);
    std::cout << std::fixed;

    // Variáveis
    int N = 100000000;  // Total de iterações
    int *device_count, *host_count;
    host_count = new int;  // Variável na CPU para receber o resultado

    // Alocando espaço na GPU para o resultado
    hipMalloc(&device_count, sizeof(int));

    // Configurando a execução com um grande número de blocos e threads por bloco
    dim3 blocks(1024, 1, 1);
    dim3 threads(1024, 1, 1);

    // Semente aleatória única para evitar resultados diferentes a cada execução
    unsigned long long seed = 12345;

    // Chamando o kernel CUDA
    monteCarloEuler<<<blocks, threads>>>(device_count, N, seed);

    // Copiando o resultado de volta para a CPU
    hipMemcpy(host_count, device_count, sizeof(int), hipMemcpyDeviceToHost);

    // Liberando a memória na GPU
    hipFree(device_count);

    // Calculando Euler
    double M_fraction = static_cast<double>(*host_count) / N;
    double euler = 1.0 / M_fraction;
    std::cout << "Euler = " << euler << std::endl;

    delete host_count;

    return 0;
}
