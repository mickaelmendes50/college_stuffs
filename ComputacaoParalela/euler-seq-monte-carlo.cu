
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void monteCarloEuler(unsigned int *count, unsigned int N) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int seed = clock() + tid;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    unsigned int local_count = 0;

    for (unsigned int i = 0; i < N; ++i) {
        double x = hiprand_uniform_double(&state);
        double y = hiprand_uniform_double(&state);

        if (x * y >= 1.0) {
            local_count++;
        }
    }

    atomicAdd(count, local_count);
}

int main() {
    unsigned int N = 1000000;  // Total de iterações
    unsigned int *d_count, h_count;

    // Alocando espaço na GPU para o resultado
    hipMalloc(&d_count, sizeof(unsigned int));
    hipMemcpy(d_count, &h_count, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Configurando a execução com uma única thread
    dim3 blocks(ceil(N / 1024.0), 1, 1);
    dim3 threads(1024, 1, 1);

    // Chamando o kernel CUDA
    monteCarloEuler<<<blocks, threads>>>(d_count, N);

    // Copiando o resultado de volta para a CPU
    hipMemcpy(&h_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Liberando a memória na GPU
    hipFree(d_count);

    // Calculando Euler
    double euler = pow(2, static_cast<double>(N) / (N - h_count));
    std::cout << "Euler = " << euler << std::endl;

    return 0;
}
