
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

__global__ void monteCarloPI(unsigned int *count, unsigned int N) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int seed = clock() + tid;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    unsigned int local_count = 0;

    for (unsigned int i = 0; i < N; ++i) {
        double x = hiprand_uniform_double(&state);
        double y = hiprand_uniform_double(&state);

        if (x*x + y*y >= 1.0) {
            local_count++;
        }
    }

    atomicAdd(count, local_count);
}

int main() {
    unsigned int N = 1000000;  // Total de iterações
    unsigned int *d_count, h_count;

    // Alocando espaço na GPU para o resultado
    hipMalloc(&d_count, sizeof(unsigned int));
    hipMemcpy(d_count, &h_count, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Configurando a execução com uma única thread
    dim3 blocks(ceil(N / 1024.0), 1, 1);
    dim3 threads(1024, 1, 1);

    // Chamando o kernel CUDA
    monteCarloPI<<<blocks, threads>>>(d_count, N);

    // Copiando o resultado de volta para a CPU
    hipMemcpy(&h_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Liberando a memória na GPU
    hipFree(d_count);

    // Calculando PI
    double pi = 4 * (N - h_count) / static_cast<double>(N);
    std::cout << "PI = " << pi << std::endl;

    return 0;
}
