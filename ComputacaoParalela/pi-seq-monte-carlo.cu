
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

__global__ void monteCarloPI(int *count, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int seed = clock() + tid;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    int local_k = 0;

    for (int i = 0; i < N; ++i) {
        // Gerando valor aleatório para os pontos de X e Y
        double x = hiprand_uniform_double(&state);
        double y = hiprand_uniform_double(&state);

        // Se o ponto não está dentro da área, incrementamos k
        if (x * x + y * y >= 1.0) {
            local_k++;
        }
    }

    atomicAdd(count, local_k);
}

int main() {
    // Configurando a precisão do resultado
    std::cout << std::setprecision(10);
	std::cout << std::fixed;

    // Variáveis
    int N = 1000000000;  // Total de iterações: 1.000.000.000
    int k = 0;
    int *device_count;

    // Alocando espaço na GPU para o resultado
    hipMalloc(&device_count, sizeof(int));
    hipMemcpy(device_count, &k, sizeof(int), hipMemcpyHostToDevice);

    // Chamando o kernel CUDA
    monteCarloPI<<<1, 1>>>(device_count, N);

    // Copiando o resultado de volta para a CPU
    hipMemcpy(&k, device_count, sizeof(int), hipMemcpyDeviceToHost);

    // Liberando a memória na GPU
    hipFree(device_count);

    // Calculando PI
    double M = N - k; // valor de pontos na área
    double pi = 4 * M / static_cast<double>(N);
    std::cout << "PI = " << pi << std::endl;

    return 0;
}
